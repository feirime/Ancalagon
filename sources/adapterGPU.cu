#include "adapterGPU.h"
#include "gpu.h"
#include <iostream>

void latticeConstructorDOSAdapter(long long int *&G, float *&E, int *&M)
{
    hipMallocManaged(&G, sizeof(G));
    hipMallocManaged(&E, sizeof(E));
    hipMallocManaged(&M, sizeof(M));
}

void latticeConstructorAdapter(float *&x, float *&y, float *&mx, float *&my, int size)
{
    hipMallocManaged(&x, size * sizeof(x));
    hipMallocManaged(&y, size * sizeof(y));
    hipMallocManaged(&mx, size * sizeof(mx));
    hipMallocManaged(&my, size * sizeof(my));
}

void latticeDestructorAdapter(long long int *&G, float *&E, int *&M, float *&x, float *&y, float *&mx, float *&my)
{
    if(my != nullptr)
    {
        hipFree(my);
        printf("free my\n");
    }
    if(mx != nullptr)
    {
        hipFree(mx);
        printf("free mx\n");
    }
    if(y != nullptr)
    {
        hipFree(y);
        printf("free y\n");
    }
    if(x != nullptr)
    {
        hipFree(x);
        printf("free x\n");
    }
    if(M != nullptr)
    {
        hipFree(M);
        printf("free M\n");
    }
    if(E != nullptr)
    {
        hipFree(E);
        printf("free E\n");
    }
    if(G != nullptr)
    {
        hipFree(G);
        printf("free G\n");
    }
}

void calculateAdapter(long long int *&G, float *&E, int *&M, float *&x, float *&y, float *&mx, float *&my, int latticeSize, float splitSeed)
{
    hipDeviceProp_t dev{};
    hipGetDeviceProperties(&dev, 0);
    static size_t block_dim = 512;
    static size_t grid_dim = get_SP_cores(dev);
    std::cout << "sp_cores: " << get_SP_cores(dev) << "\n";
    unsigned int *configuration;
    //mapMaker(x, y, mx, my, latticeSize, splitSeed);
    //unifing<<<grid_dim, block_dim>>>();
    hipDeviceSynchronize();
}
