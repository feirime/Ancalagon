#include "adapterGPU.h"
#include "gpu.h"
#include <iostream>

void latticeConstructorAdapter(long long int *&G, float *&E, int *&M)
{
    printf("G = %p, E = %p, M = %p\n", G, E, M);
    hipMallocManaged(&G, sizeof(G));
    hipMallocManaged(&E, sizeof(E));
    hipMallocManaged(&M, sizeof(M));
    printf("G = %p, E = %p, M = %p\n", G, E, M);
}

void latticeDestructorAdapter(long long int *&G, float *&E, int *&M)
{
    if(G != nullptr)
        hipFree(G);
    if(E != nullptr)
        hipFree(E);
    if(M != nullptr)
        hipFree(M);
}

void calculateAdapter(long long int *&G, float *&E, int *&M)
{
    printf("test CPU\n");
    test<<<1, 1>>>();
    hipDeviceSynchronize();
}
