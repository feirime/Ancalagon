#include "adapterGPU.h"
#include "gpu.h"
#include <iostream>

void latticeConstructorAdapter(long long int *&G, float *&E, int *&M)
{
    hipMallocManaged(&G, sizeof(G));
    hipMallocManaged(&E, sizeof(E));
    hipMallocManaged(&M, sizeof(M));
}

void latticeDestructorAdapter(long long int *&G, float *&E, int *&M)
{
    if(G != nullptr)
        hipFree(G);
        printf("free G\n");
    if(E != nullptr)
        hipFree(E);
        printf("free E\n");
    if(M != nullptr)
        hipFree(M);
        printf("free M\n");
}

void calculateAdapter(long long int *&G, float *&E, int *&M)
{
    hipDeviceProp_t dev{};
    hipGetDeviceProperties(&dev, 0);
    static size_t block_dim = 512;
    static size_t grid_dim = get_SP_cores(dev);
    std::cout << "sp_cores: " << get_SP_cores(dev) << "\n";
    hipDeviceSynchronize();
}
