#include "adapterGPU.h"
#include "gpu.h"
#include <iostream>

void latticeConstructorDOSAdapter(long long int *&G, float *&E, int *&M, int size)
{
    hipMallocManaged(&G, size * sizeof(G));
    hipMallocManaged(&E, size * sizeof(E));
    hipMallocManaged(&M, size * sizeof(M));
}

void latticeConstructorAdapter(float *&x, float *&y, float *&mx, float *&my, int size)
{
    hipMallocManaged(&x, size * sizeof(x));
    hipMallocManaged(&y, size * sizeof(y));
    hipMallocManaged(&mx, size * sizeof(mx));
    hipMallocManaged(&my, size * sizeof(my));
}

void latticeDestructorAdapter(long long int *&Geven, float *&Eeven, int *&Meven, 
    long long int *&Godd, float *&Eodd, int *&Modd,  float *&x, float *&y, float *&mx, float *&my)
{
    if(my != nullptr)
    {
        hipFree(my);
        printf("free my\n");
    }
    if(mx != nullptr)
    {
        hipFree(mx);
        printf("free mx\n");
    }
    if(y != nullptr)
    {
        hipFree(y);
        printf("free y\n");
    }
    if(x != nullptr)
    {
        hipFree(x);
        printf("free x\n");
    }
    if(Meven != nullptr)
    {
        hipFree(Meven);
        printf("free Meven\n");
    }
    if(Eeven != nullptr)
    {
        hipFree(Eeven);
        printf("free Eeven\n");
    }
    if(Geven != nullptr)
    {
        hipFree(Geven);
        printf("free Geven\n");
    }
    if(Modd != nullptr)
    {
        hipFree(Modd);
        printf("free Modd\n");
    }
    if(Eodd != nullptr)
    {
        hipFree(Eodd);
        printf("free Eodd\n");
    }
    if(Godd != nullptr)
    {
        hipFree(Godd);
        printf("free Godd\n");
    }
}

void calculateAdapter(long long int *&G, float *&E, int *&M, float *&x, float *&y, float *&mx, float *&my, int latticeSize, float splitSeed)
{
    hipDeviceProp_t dev{};
    hipGetDeviceProperties(&dev, 0);
    static size_t block_dim = 512;
    static size_t grid_dim = get_SP_cores(dev);
    std::cout << "sp_cores: " << get_SP_cores(dev) << "\n";
    unsigned int *configuration;
    //mapMaker(x, y, mx, my, latticeSize, splitSeed);
    //unifing<<<grid_dim, block_dim>>>();
    hipDeviceSynchronize();
}
