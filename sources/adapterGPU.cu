#include "adapterGPU.h"
#include "gpu.h"
#include <iostream>

void latticeConstructorAdapter(long long int *&G, float *&E, int *&M, float *&x, float *&y, float *&mx, float *&my, int linearSize)
{
    hipMallocManaged(&G, sizeof(G));
    hipMallocManaged(&E, sizeof(E));
    hipMallocManaged(&M, sizeof(M));
    hipMallocManaged(&x, linearSize * sizeof(x));
    hipMallocManaged(&y, linearSize * sizeof(y));
    hipMallocManaged(&mx, linearSize * sizeof(mx));
    hipMallocManaged(&my, linearSize * sizeof(my));
}

void latticeDestructorAdapter(long long int *&G, float *&E, int *&M, float *&x, float *&y, float *&mx, float *&my)
{
    if(my != nullptr)
    {
        hipFree(my);
        printf("free my\n");
    }
    if(mx != nullptr)
    {
        hipFree(mx);
        printf("free mx\n");
    }
    if(y != nullptr)
    {
        hipFree(y);
        printf("free y\n");
    }
    if(x != nullptr)
    {
        hipFree(x);
        printf("free x\n");
    }
    if(G != nullptr)
    {
        hipFree(G);
        printf("free G\n");
    }
    if(E != nullptr)
    {
        hipFree(E);
        printf("free E\n");
    }
    if(M != nullptr)
    {
        hipFree(M);
        printf("free M\n");
    }
}

void calculateAdapter(long long int *&G, float *&E, int *&M)
{
    hipDeviceProp_t dev{};
    hipGetDeviceProperties(&dev, 0);
    static size_t block_dim = 512;
    static size_t grid_dim = get_SP_cores(dev);
    std::cout << "sp_cores: " << get_SP_cores(dev) << "\n";
    hipDeviceSynchronize();
}
