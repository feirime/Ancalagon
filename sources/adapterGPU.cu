#include "adapterGPU.h"
#include "gpu.h"
#include <iostream>

void testAdapterGPU()
{
    printf("test CPU\n");
    test<<<1, 1>>>();
    hipDeviceSynchronize();
}

void latticeConstructor(long long int *G, float *E, int *M)
{
    hipMallocManaged(&G, sizeof(G));
    hipMallocManaged(&E, sizeof(E));
    hipMallocManaged(&M, sizeof(M));
}

void latticeDestructor(long long int *G, float *E, int *M)
{
    hipFree(G);
    hipFree(E);
    hipFree(M);
}
