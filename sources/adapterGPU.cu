#include "adapterGPU.h"
#include "gpu.h"
#include <iostream>

void latticeConstructorDOSAdapter(long long int *&G, float *&E, int *&M, int size)
{
    hipMallocManaged(&G, size * sizeof(G));
    hipMallocManaged(&E, size * sizeof(E));
    hipMallocManaged(&M, size * sizeof(M));
}

void latticeConstructorAdapter(float *&x, float *&y, float *&mx, float *&my, int size)
{
    hipMallocManaged(&x, size * sizeof(x));
    hipMallocManaged(&y, size * sizeof(y));
    hipMallocManaged(&mx, size * sizeof(mx));
    hipMallocManaged(&my, size * sizeof(my));
}

void latticeDestructorAdapter(long long int *&Gmain, float *&Emain, int *&Mmain, 
    long long int *&Gresult, float *&Eresult, int *&Mresult,  float *&x, float *&y, float *&mx, float *&my)
{
    if(my != nullptr)
    {
        hipFree(my);
        printf("free my\n");
    }
    if(mx != nullptr)
    {
        hipFree(mx);
        printf("free mx\n");
    }
    if(y != nullptr)
    {
        hipFree(y);
        printf("free y\n");
    }
    if(x != nullptr)
    {
        hipFree(x);
        printf("free x\n");
    }
    if(Mmain != nullptr)
    {
        hipFree(Mmain);
        printf("free Mmain\n");
    }
    if(Emain != nullptr)
    {
        hipFree(Emain);
        printf("free Emain\n");
    }
    if(Gmain != nullptr)
    {
        hipFree(Gmain);
        printf("free Gmain\n");
    }
    if(Mresult != nullptr)
    {
        hipFree(Mresult);
        printf("free Mresult\n");
    }
    if(Eresult != nullptr)
    {
        hipFree(Eresult);
        printf("free Eresult\n");
    }
    if(Gresult != nullptr)
    {
        hipFree(Gresult);
        printf("free Gresult\n");
    }
}

void calculateAdapter(long long int *&G, float *&E, int *&M, float *&x, float *&y, float *&mx, float *&my, int latticeSize, float splitSeed)
{
    hipDeviceProp_t dev{};
    hipGetDeviceProperties(&dev, 0);
    static size_t block_dim = 512;
    static size_t grid_dim = get_SP_cores(dev);
    std::cout << "sp_cores: " << get_SP_cores(dev) << "\n";
    unsigned int *configuration;
    //mapMaker(x, y, mx, my, latticeSize, splitSeed);
    //unifing<<<grid_dim, block_dim>>>();
    hipDeviceSynchronize();
}
