#include "hip/hip_runtime.h"

#include <iostream>
#include "adapterGPU.h"
#include "gpu.h"

__global__ void unifingSquare(int rightLayer, Cell *left, int leftSize, Cell *right, int rightSize, Cell *result, float *JMap) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    for(int i = 0; i < leftSize; i++)
    {
        for(int j = 0; j < rightSize; j++)
        {
            result[j].M = left[i].M + right[j].M;
            for(int bound = 0; bound < 1 + rightLayer * 3; bound++)
            {
                result[j].E = left[i].E + right[j].E + JMap[i];
            }
            result[j].G = left[i].G;
        }
    }
}

void testAdapter()
{
    printf("test CPU\n");
    test<<<1, 1>>>();
    hipDeviceSynchronize();
}

__global__ void test()
{
    printf("test GPU\n");
}