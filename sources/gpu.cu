#include "hip/hip_runtime.h"

#include <iostream>
#include "adapterGPU.h"
#include "gpu.h"

int get_SP_cores(hipDeviceProp_t devProp)
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 7: // Volta and Turing
            if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 8: // Ampere
            if (devProp.minor == 0) cores = mp * 64;
            else if (devProp.minor == 6) cores = mp * 128;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

__global__ void test()
{
    if(threadIdx.x == 0 && blockIdx.x == 0)
        printf("test GPU\n");
}

void mapMaker(double *x, double *y, double *mx, double *my, int latticeSize, float seed)
{
    double xMin = *std::min_element(x, x + latticeSize);
    double xMax = *std::max_element(x, x + latticeSize);
    double yMin = *std::min_element(y, y + latticeSize);
    double yMax = *std::max_element(y, y + latticeSize);

    for(auto i = 0; i < latticeSize; i++)
    {
        int j = 0;
        double xPrevious = x[j];
        while(x[j] == xPrevious)
        {
            j++;
        }
    }
}

__global__ void calculate(long long int *G, float *E, int *M, double *x, double *y, double *mx, double *my, int latticeSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
}
